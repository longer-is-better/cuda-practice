#include<hip/hip_runtime.h>
#include"hip/hip_runtime_api.h"
#include"fun.cuh"

int main(){
    dim3 grid(1, 1, 2), block(1, 2, 1);
    fun<<<grid, block>>>();
    checkCudaErrors(hipDeviceSynchronize());
}