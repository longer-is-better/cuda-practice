#include "hip/hip_runtime.h"
#include<string>
#include<string.h>
#include<vector>
#include<iostream>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime_api.h"
#include"descriptor.h"
#include"log.h"


TensorDesc::TensorDesc(const std::string& layout, const std::vector<int>& shape) {
    if (layout.size() != shape.size()) LOGERR("layout.size() != shape.size()");

    checkCudaErrors(hipMallocManaged((void**)&this->dim_n, sizeof(int)));
    *dim_n = layout.size();

    checkCudaErrors(hipMallocManaged((void**)&this->layout, layout.size() + 1));
    memcpy(this->layout, layout.c_str(), layout.size() + 1);

    checkCudaErrors(hipMallocManaged((void**)&this->shape, shape.size() * sizeof(int)));
    memcpy(this->shape, shape.data(), shape.size() * sizeof(int));

    checkCudaErrors(hipMallocManaged((void**)&this->stride, shape.size() * sizeof(int)));
    this->stride[shape.size() - 1] = 1;
    for (int i = shape.size() - 1; i > 0; i--) this->stride[i - 1] = this->stride[i] * this->shape[i];
}

TensorDesc::~TensorDesc(){
    checkCudaErrors(hipFree(dim_n));
    checkCudaErrors(hipFree(layout));
    checkCudaErrors(hipFree(shape));
    checkCudaErrors(hipFree(stride));
}

void TensorDesc::init(const std::string& layout, const std::vector<int>& shape) {
    if (layout.size() != shape.size()) LOGERR("layout.size() != shape.size()");

    checkCudaErrors(hipMallocManaged((void**)&this->dim_n, sizeof(int)));
    *dim_n = layout.size();

    checkCudaErrors(hipMallocManaged((void**)&this->layout, layout.size() + 1));
    memcpy(this->layout, layout.c_str(), layout.size() + 1);

    checkCudaErrors(hipMallocManaged((void**)&this->shape, shape.size() * sizeof(int)));
    memcpy(this->shape, shape.data(), shape.size() * sizeof(int));

    checkCudaErrors(hipMallocManaged((void**)&this->stride, shape.size() * sizeof(int)));
    this->stride[shape.size() - 1] = 1;
    for (int i = shape.size() - 1; i > 0; i--) this->stride[i - 1] = this->stride[i] * this->shape[i];
}

void* TensorDesc::operator new(std::size_t size) {
    void* ptr = nullptr;
    checkCudaErrors(hipMallocManaged(&ptr, size));
    return ptr;
}

void TensorDesc::operator delete(void* ptr) {
    checkCudaErrors(hipFree(ptr));
}

TensorDesc::TensorDesc(TensorDesc&& rvalue){
    std::cout << "move TensorDesc, yes!" << std::endl;
    this->dim_n = rvalue.dim_n;
    rvalue.dim_n = 0;
    this->layout = rvalue.layout; rvalue.layout = nullptr;
    this->shape = rvalue.shape; rvalue.shape = nullptr;
    this->stride = rvalue.stride; rvalue.stride = nullptr;
}

TensorDesc& TensorDesc::operator=(TensorDesc&& rvalue){
    std::cout << "move assign TensorDesc, yes!" << std::endl;
    this->dim_n = rvalue.dim_n; rvalue.dim_n = 0;
    this->layout = rvalue.layout; rvalue.layout = nullptr;
    this->shape = rvalue.shape; rvalue.shape = nullptr;
    this->stride = rvalue.stride; rvalue.stride = nullptr;
    return *this;
};

Pool2dDesc::Pool2dDesc(const std::vector<int>& window_shape, const std::vector<int>& padding, const std::vector<int>& stride) {
    if (window_shape.size() != stride.size()) LOGERR("window_shape.size() != stride.size()");
    
    checkCudaErrors(hipMallocManaged((void**)&this->dim_n, sizeof(int)));
    *dim_n = window_shape.size();

    checkCudaErrors(hipMallocManaged((void**)&this->window_shape, window_shape.size() * sizeof(int)));
    memcpy(this->window_shape, window_shape.data(), window_shape.size() *  sizeof(int));

    checkCudaErrors(hipMallocManaged((void**)&this->stride, stride.size() * sizeof(int)));
    memcpy(this->stride, stride.data(), stride.size() *  sizeof(int));

    checkCudaErrors(hipMallocManaged((void**)&this->padding, padding.size() * sizeof(int)));
    memcpy(this->padding, padding.data(), padding.size() *  sizeof(int));
}

Pool2dDesc& Pool2dDesc::operator=(Pool2dDesc&& rv) {
    this->dim_n = rv.dim_n; rv.dim_n = 0;

    this->window_shape = rv.window_shape; rv.window_shape = nullptr;
    this->stride = rv.stride; rv.stride = nullptr;
    this->padding = rv.padding; rv.padding = nullptr;

    return *this;
}

Pool2dDesc::~Pool2dDesc() {
    checkCudaErrors(hipFree(dim_n));
    checkCudaErrors(hipFree(window_shape));
    checkCudaErrors(hipFree(stride));
    checkCudaErrors(hipFree(padding));
}

void* Pool2dDesc::operator new(std::size_t size) {
    void* ptr = nullptr;
    checkCudaErrors(hipMallocManaged(&ptr, size));
    return ptr;
}

void Pool2dDesc::operator delete(void* ptr) {
    checkCudaErrors(hipFree(ptr));
}
