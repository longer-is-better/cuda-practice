#include "hip/hip_runtime.h"
#include<string>
#include<string.h>
#include<vector>
#include<iostream>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime_api.h"
#include"descriptor.h"
#include"log.h"


TensorDesc::TensorDesc(const std::string& layout, const std::vector<int>& shape) {
    if (layout.size() != shape.size()) LOGERR("layout.size() != shape.size()");

    checkCudaErrors(hipMallocManaged((void**)&this->dim_n, sizeof(int)));
    *dim_n = layout.size();

    checkCudaErrors(hipMallocManaged((void**)&this->layout, layout.size() + 1));
    memcpy(this->layout, layout.c_str(), layout.size() + 1);

    checkCudaErrors(hipMallocManaged((void**)&this->shape, shape.size() * sizeof(int)));
    memcpy(this->shape, shape.data(), shape.size() * sizeof(int));

    checkCudaErrors(hipMallocManaged((void**)&this->stride, shape.size() * sizeof(int)));
    this->stride[shape.size() - 1] = 1;
    for (int i = shape.size() - 1; i > 0; i--) this->stride[i - 1] = this->stride[i] * this->shape[i];
}

TensorDesc::~TensorDesc(){
    hipFree(dim_n);
    hipFree(layout);
    hipFree(shape);
    hipFree(stride);
}

void TensorDesc::init(const std::string& layout, const std::vector<int>& shape) {
    if (layout.size() != shape.size()) LOGERR("layout.size() != shape.size()");

    checkCudaErrors(hipMallocManaged((void**)&this->dim_n, sizeof(int)));
    *dim_n = layout.size();

    checkCudaErrors(hipMallocManaged((void**)&this->layout, layout.size() + 1));
    memcpy(this->layout, layout.c_str(), layout.size() + 1);

    checkCudaErrors(hipMallocManaged((void**)&this->shape, shape.size() * sizeof(int)));
    memcpy(this->shape, shape.data(), shape.size() * sizeof(int));

    checkCudaErrors(hipMallocManaged((void**)&this->stride, shape.size() * sizeof(int)));
    this->stride[shape.size() - 1] = 1;
    for (int i = shape.size() - 1; i > 0; i--) this->stride[i - 1] = this->stride[i] * this->shape[i];
}

void* TensorDesc::operator new(std::size_t size) {
    void* ptr = nullptr;
    checkCudaErrors(hipMallocManaged(&ptr, size));
    return ptr;
}

void TensorDesc::operator delete(void* ptr) {
    checkCudaErrors(hipFree(ptr));
}

TensorDesc::TensorDesc(TensorDesc&& rvalue){
    std::cout << "move TensorDesc, yes!" << std::endl;
    this->dim_n = rvalue.dim_n;
    rvalue.dim_n = 0;
    this->layout = rvalue.layout; rvalue.layout = nullptr;
    this->shape = rvalue.shape; rvalue.shape = nullptr;
    this->stride = rvalue.stride; rvalue.stride = nullptr;
}

TensorDesc& TensorDesc::operator=(TensorDesc&& rvalue){
    std::cout << "move assign TensorDesc, yes!" << std::endl;
    this->dim_n = rvalue.dim_n; rvalue.dim_n = 0;
    this->layout = rvalue.layout; rvalue.layout = nullptr;
    this->shape = rvalue.shape; rvalue.shape = nullptr;
    this->stride = rvalue.stride; rvalue.stride = nullptr;
    return *this;
};