#include<string>
#include<string.h>
#include<vector>
#include<iostream>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime_api.h"
#include"descriptor.h"
#include"log_err.h"


TensorDesc::TensorDesc(const std::string& layout, const std::vector<int>& shape) {
    if (layout.size() != shape.size()) LOGERR("layout.size() != shape.size()");

    checkCudaErrors(hipMallocManaged((void**)&this->dim_n, sizeof(int)));
    *dim_n = layout.size();

    checkCudaErrors(hipMallocManaged((void**)&this->layout, layout.size() + 1));
    memcpy(this->layout, layout.c_str(), layout.size() + 1);

    checkCudaErrors(hipMallocManaged((void**)&this->shape, shape.size() * sizeof(int)));
    memcpy(this->shape, shape.data(), shape.size() * sizeof(int));
}

TensorDesc::~TensorDesc(){
    free(layout);
    hipFree(shape);
}

void TensorDesc::init(const std::string& layout, const std::vector<int>& shape) {
    if (layout.size() != shape.size()) LOGERR("layout.size() != shape.size()");

    checkCudaErrors(hipMallocManaged((void**)&this->dim_n, sizeof(int)));
    *dim_n = layout.size();

    checkCudaErrors(hipMallocManaged((void**)&this->layout, layout.size() + 1));
    memcpy(this->layout, layout.c_str(), layout.size() + 1);

    checkCudaErrors(hipMallocManaged((void**)&this->shape, shape.size() * sizeof(int)));
    memcpy(this->shape, shape.data(), shape.size() * sizeof(int));
}




TensorDesc::TensorDesc(TensorDesc&& rvalue){
    std::cout << "move TensorDesc, yes!" << std::endl;
    this->dim_n = rvalue.dim_n;
    rvalue.dim_n = 0;
    this->layout = rvalue.layout; rvalue.layout = nullptr;
    this->shape = rvalue.shape; rvalue.shape = nullptr;
}

TensorDesc& TensorDesc::operator=(TensorDesc&& rvalue){
    std::cout << "move assign TensorDesc, yes!" << std::endl;
    this->dim_n = rvalue.dim_n; rvalue.dim_n = 0;
    this->layout = rvalue.layout; rvalue.layout = nullptr;
    this->shape = rvalue.shape; rvalue.shape = nullptr;
    return *this;
};